#include "hip/hip_runtime.h"
#include "lb.h"
#include "common.h"

double error()
{
	int q, k;
	double ut, vt, wt;
	double sum_u_c = 0.0, sum_v_c = 0.0, sum_w_c = 0.0;
	double Err;
	FILE *fp;

	for(k = 0; k < N; k++)
	{
		ut = vt = wt = 0.0;
		for(q = 1; q < Q; q++)
		{
			ut += e[q][0]*f[k+q*size];
			vt += e[q][1]*f[k+q*size];
			wt += e[q][2]*f[k+q*size];

		}	
		ut += 0.5*dt*Fx;
		vt += 0.5*dt*Fy;
		wt += 0.5*dt*Fz;

		sum_u_c += ut;
		sum_v_c += vt;
		sum_w_c += wt;
	}
	

//	Err = fabs(sum_u_c - sum_u_o)/fabs(sum_u_c + 1.0e-9) + fabs(sum_v_c - sum_v_o)/fabs(sum_v_c + 1.0e-9) + fabs(sum_w_c - sum_w_o)/fabs(sum_w_c + 1.0e-9);
	Err = fabs(sum_u_c - sum_u_o)/fabs(sum_u_c + 1.0e-9);
	sum_u_o = sum_u_c;
	sum_v_o = sum_v_c;
	sum_w_o = sum_w_c;

	if((fp=fopen("err.dat", "a"))==NULL) printf("FILE OPEN ERROR!\n");
	fprintf(fp, "%d %e\n", n, Err);
	fclose(fp);

	return Err;

}
