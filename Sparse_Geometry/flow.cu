#include "hip/hip_runtime.h"
#include "lb.h"
#include "common.h"

//used for MRT-LBE model
__constant__ double r0 = 1.0/19;
__constant__ double r1 = 5.0/399;
__constant__ double r2 = 1.0/21;
__constant__ double r3 = 11.0/2394;
__constant__ double r4 = 1.0/63;
__constant__ double r5 = 1.0/10;
__constant__ double r6 = 1.0/18;
__constant__ double r7 = 1.0/36;
__constant__ double r8 = 4.0/1197;
__constant__ double r9 = 1.0/252;
__constant__ double r10 = 1.0/40;
__constant__ double r11 = 1.0/12;
__constant__ double r12 = 1.0/4;
__constant__ double r13 = 1.0/72;
__constant__ double r14 = 1.0/24;
__constant__ double r15 = 1.0/8;
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Evol_flow(double rgama, double *sf_d, double dt, double Fx, double Fy, double Fz, int N, int size, int *node_d, double *f_d, double *F_d)
{
	double P, U, V, W;
	int tx;
	int bx;
	int k;	
	
	double mf[Q];
	double f0, f1, f2, f3, f4, f5, f6, f7, f8, f9, f10, f11, f12, f13, f14, f15, f16, f17, f18;
	double F0, F1, F2, F3, F4, F5, F6, F7, F8, F9, F10, F11, F12, F13, F14, F15, F16, F17, F18;
	int Id_0, Id_1, Id_2, Id_3, Id_4, Id_5, Id_6, Id_7, Id_8, Id_9, Id_10, Id_11, Id_12, Id_13, Id_14, Id_15, Id_16, Id_17, Id_18;

	tx = threadIdx.x;
    bx = blockIdx.x;
	k = bx*BX+tx;


	if(k < N)
	{

		f0  = f_d[k+0*size]; 
		f1  = f_d[k+1*size]; 
		f2  = f_d[k+2*size]; 
		f3  = f_d[k+3*size]; 
		f4  = f_d[k+4*size]; 
		f5  = f_d[k+5*size]; 
		f6  = f_d[k+6*size]; 
		f7  = f_d[k+7*size]; 
		f8  = f_d[k+8*size]; 
		f9  = f_d[k+9*size];

		f10 = f_d[k+10*size]; 
		f11 = f_d[k+11*size]; 
		f12 = f_d[k+12*size]; 
		f13 = f_d[k+13*size]; 
		f14 = f_d[k+14*size]; 
		f15 = f_d[k+15*size]; 
		f16 = f_d[k+16*size]; 
		f17 = f_d[k+17*size]; 
		f18 = f_d[k+18*size];

		Id_0 = node_d[k + 0*size];
		Id_1 = node_d[k + 1*size];
		Id_2 = node_d[k + 2*size];
		Id_3 = node_d[k + 3*size];
		Id_4 = node_d[k + 4*size];
		Id_5 = node_d[k + 5*size];
		Id_6 = node_d[k + 6*size];
		Id_7 = node_d[k + 7*size];
		Id_8 = node_d[k + 8*size];
		Id_9 = node_d[k + 9*size];

		Id_10 = node_d[k + 10*size];
		Id_11 = node_d[k + 11*size];
		Id_12 = node_d[k + 12*size];
		Id_13 = node_d[k + 13*size];
		Id_14 = node_d[k + 14*size];
		Id_15 = node_d[k + 15*size];
		Id_16 = node_d[k + 16*size];
		Id_17 = node_d[k + 17*size];
		Id_18 = node_d[k + 18*size];
		
		
		//f-mf///////////////////////////
		mf[0]  = f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8 + f9 + f10 + f11 + f12 + f13 + f14 + f15 + f16 + f17 + f18; 
 		mf[1]  = -30.0*f0 - 11.0*f1 - 11.0*f2 - 11.0*f3 - 11.0*f4 - 11.0*f5 -11.0*f6 + 8.0*f7 + 8.0*f8 + 8.0*f9 
				+ 8.0*f10 + 8.0*f11 + 8.0*f12 + 8.0*f13 + 8.0*f14 + 8.0*f15 + 8.0*f16 + 8.0*f17 + 8.0*f18;
        mf[2]  = 12.0*f0 - 4.0*f1 - 4.0*f2 - 4.0*f3 - 4.0*f4 - 4.0*f5 - 4.0*f6 + f7 + f8 + f9 
				+ f10 + f11 + f12 + f13 + f14 + f15 + f16 + f17 + f18;
        mf[3]  = f1 - f2 - f11 + f12 - f13 + f14 - f15 + f16 - f17 + f18;
        mf[4]  = -4.0*f1 + 4.0*f2 - f11 + f12 - f13 + f14 - f15 + f16 - f17 + f18;
        mf[5]  = f3 - f4 + f7 - f8 - f9 + f10 + f15 - f16 - f17 + f18;
        mf[6]  = -4.0*f3 + 4.0*f4 + f7 - f8 - f9 + f10 + f15 - f16 - f17 + f18;
        mf[7]  = f5 - f6 + f7 - f8 + f9 + f12 - f11 - f10 + f13 - f14;
        mf[8]  = f12 - f11 - f10 + f13 - f14 - 4.0*f5 + 4.0*f6 + f7 - f8 + f9;
        mf[9]  = 2.0*f1 - 2.0*f10 + f11 + f12 + f13 + f14 + f15 + f16 
				+ f17 + f18 + 2.0*f2 - f3 - f4 - f5 - f6 - 2.0*f7 - 2.0*f8 - 2.0*f9;

        mf[10] = f11 - 2.0*f10 - 4.0*f1 + f12 + f13 + f14 + f15 
				+ f16 + f17 + f18 - 4.0*f2 + 2.0*f3 + 2.0*f4 + 2.0*f5 + 2.0*f6 - 2.0*f7 - 2.0*f8 - 2.0*f9;
        mf[11] = f15 - f12 - f13 - f14 - f11 + f16 + f17 + f18 + f3 + f4 - f5 - f6;
        mf[12] = f15 - f12 - f13 - f14 - f11 + f16 + f17 + f18 - 2.0*f3 - 2.0*f4 + 2.0*f5 + 2.0*f6;
        mf[13] = f17 - f16 - f15 + f18;
        mf[14] = f7 - f10 + f8 - f9;
        mf[15] = f11 + f12 - f13 - f14;
        mf[16] = f11 - f12 + f13 - f14 - f15 + f16 - f17 + f18;
        mf[17] = f10 - f15 + f16 + f17 - f18 + f7 - f8 - f9;
        mf[18] = f10 - f11 + f12 + f13 - f14 - f7 + f8 - f9;

		//macroscopic variables/////////////////////////////////////////////////////////////////////////////
		U   = f1 + f12 + f14 + f16 + f18 - f2 - f11 - f13 - f15 - f17 + 0.5*dt*Fx;
		V   = f3 + f7  + f10 + f15 + f18 - f4 - f8  - f9  - f16 - f17 + 0.5*dt*Fy;
		W   = f5 + f7  + f9  + f12 + f13 - f6 - f8  - f10 - f11 - f14 + 0.5*dt*Fz;
		P = (f0 + f1 + f2 + f3 + f4 + f5 + f6 + f7 + f8 + f9 + f10 + f11 + f12 + f13 + f14 + f15 + f16 + f17 + f18);
		
		//collision//-------------------------------------------------------------------------------------------------------------------------------
		mf[0]  = (mf[0] - sf_d[0]*(mf[0] - MEQ_0(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[0])*F_0(U,V,W,Fx,Fy,Fz,rgama);
		mf[1]  = (mf[1] - sf_d[1]*(mf[1] - MEQ_1(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[1])*F_1(U,V,W,Fx,Fy,Fz,rgama);
		mf[2]  = (mf[2] - sf_d[2]*(mf[2] - MEQ_2(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[2])*F_2(U,V,W,Fx,Fy,Fz,rgama);
		mf[3]  = (mf[3] - sf_d[3]*(mf[3] - MEQ_3(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[3])*F_3(U,V,W,Fx,Fy,Fz,rgama);
		mf[4]  = (mf[4] - sf_d[4]*(mf[4] - MEQ_4(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[4])*F_4(U,V,W,Fx,Fy,Fz,rgama);
		mf[5]  = (mf[5] - sf_d[5]*(mf[5] - MEQ_5(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[5])*F_5(U,V,W,Fx,Fy,Fz,rgama);
		mf[6]  = (mf[6] - sf_d[6]*(mf[6] - MEQ_6(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[6])*F_6(U,V,W,Fx,Fy,Fz,rgama);
		mf[7]  = (mf[7] - sf_d[7]*(mf[7] - MEQ_7(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[7])*F_7(U,V,W,Fx,Fy,Fz,rgama);
		mf[8]  = (mf[8] - sf_d[8]*(mf[8] - MEQ_8(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[8])*F_8(U,V,W,Fx,Fy,Fz,rgama);
		mf[9]  = (mf[9] - sf_d[9]*(mf[9] - MEQ_9(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[9])*F_9(U,V,W,Fx,Fy,Fz,rgama);

		mf[10] = (mf[10]-sf_d[10]*(mf[10] -MEQ_10(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[10])*F_10(U,V,W,Fx,Fy,Fz,rgama);
		mf[11] = (mf[11]-sf_d[11]*(mf[11] -MEQ_11(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[11])*F_11(U,V,W,Fx,Fy,Fz,rgama);
		mf[12] = (mf[12]-sf_d[12]*(mf[12] -MEQ_12(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[12])*F_12(U,V,W,Fx,Fy,Fz,rgama);
		mf[13] = (mf[13]-sf_d[13]*(mf[13] -MEQ_13(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[13])*F_13(U,V,W,Fx,Fy,Fz,rgama);
		mf[14] = (mf[14]-sf_d[14]*(mf[14] -MEQ_14(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[14])*F_14(U,V,W,Fx,Fy,Fz,rgama);
		mf[15] = (mf[15]-sf_d[15]*(mf[15] -MEQ_15(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[15])*F_15(U,V,W,Fx,Fy,Fz,rgama);
		mf[16] = (mf[16]-sf_d[16]*(mf[16] -MEQ_16(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[16])*F_16(U,V,W,Fx,Fy,Fz,rgama);
		mf[17] = (mf[17]-sf_d[17]*(mf[17] -MEQ_17(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[17])*F_17(U,V,W,Fx,Fy,Fz,rgama);
		mf[18] = (mf[18]-sf_d[18]*(mf[18] -MEQ_18(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[18])*F_18(U,V,W,Fx,Fy,Fz,rgama);

		//--mf - f --//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		F0  = r0*mf[0] - r1*mf[1] + r2*mf[2];
        F1  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[3] - r5*mf[4] + r6*mf[9] - r6*mf[10];
        F2  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[3] + r5*mf[4] + r6*mf[9] - r6*mf[10]; 
        F3  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[5] - r5*mf[6] - r7*mf[9] + r7*mf[10] + r11*mf[11] - r11*mf[12]; 
        F4  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[5] + r5*mf[6] - r7*mf[9] + r7*mf[10] + r11*mf[11] - r11*mf[12];
        F5  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[7] - r5*mf[8] - r7*mf[9] + r7*mf[10] - r11*mf[11] + r11*mf[12];
        F6  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[7] + r5*mf[8] - r7*mf[9] + r7*mf[10] - r11*mf[11] + r11*mf[12];
        F7  = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[5] + r10*mf[6] + r5*mf[7] + r10*mf[8] - r6*mf[9]  - r7*mf[10] + r12*mf[14] + r15*mf[17] - r15*mf[18];
        F8  = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[5] - r10*mf[6] - r5*mf[7] - r10*mf[8] - r6*mf[9]  - r7*mf[10] + r12*mf[14] - r15*mf[17] + r15*mf[18];
        F9  = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[5] - r10*mf[6] + r5*mf[7] + r10*mf[8] - r6*mf[9]  - r7*mf[10] - r12*mf[14] - r15*mf[17] - r15*mf[18];

        F10 = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[5] + r10*mf[6] - r5*mf[7] - r10*mf[8] - r6*mf[9]  - r7*mf[10] - r12*mf[14] + r15*mf[17] + r15*mf[18];
        F11 = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] - r5*mf[7] - r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] + r12*mf[15] + r15*mf[16] - r15*mf[18];
        F12 = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] + r5*mf[7] + r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] + r12*mf[15] - r15*mf[16] + r15*mf[18];
        F13 = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] + r5*mf[7] + r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] - r12*mf[15] + r15*mf[16] + r15*mf[18]; 
        F14 = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] - r5*mf[7] - r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] - r12*mf[15] - r15*mf[16] - r15*mf[18]; 
        F15 = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] + r5*mf[5] + r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] - r12*mf[13] - r15*mf[16] - r15*mf[17]; 
        F16 = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] - r5*mf[5] - r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] - r12*mf[13] + r15*mf[16] + r15*mf[17]; 
        F17 = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] - r5*mf[5] - r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] + r12*mf[13] - r15*mf[16] + r15*mf[17];
        F18 = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] + r5*mf[5] + r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] + r12*mf[13] + r15*mf[16] - r15*mf[17];
		
		//  streaming  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		F_d[Id_0] = F0;
		F_d[Id_1] = F1;
		F_d[Id_2] = F2;
		F_d[Id_3] = F3;
		F_d[Id_4] = F4;
		F_d[Id_5] = F5;
		F_d[Id_6] = F6;
		F_d[Id_7] = F7;
		F_d[Id_8] = F8;
		F_d[Id_9] = F9;

		F_d[Id_10] = F10;
		F_d[Id_11] = F11;
		F_d[Id_12] = F12;
		F_d[Id_13] = F13;
		F_d[Id_14] = F14;
		F_d[Id_15] = F15;
		F_d[Id_16] = F16;
		F_d[Id_17] = F17;
		F_d[Id_18] = F18;
	
	}
}
