#include <stdlib.h>
#include <malloc.h>
#include "common.h"
#include "lb.h"

void memalloc()
{
	FILE *fp;
	
	if((fp=fopen("node_index", "r"))==NULL) printf("FILE OPEN ERROR!\n");
	fscanf(fp, "%d ", &N);
	fclose(fp);

	printf("Number of fluid node is %d\n", N);
    
	size = N;
	//allocate memeory at host
    f    = (double*)calloc(size*Q, sizeof(double));
    sf   = (double*)calloc(Q, sizeof(double));
    node_index = (int*)calloc(size*Q, sizeof(int));

    //allocate memeory at device
    hipMalloc((void **)&f_dev, size*Q*sizeof(double));
    hipMalloc((void **)&F_dev, size*Q*sizeof(double));
    hipMalloc((void **)&sf_dev, Q*sizeof(double));
    hipMalloc((void **)&node_dev, size*Q*sizeof(int));
}

