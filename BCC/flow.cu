#include "hip/hip_runtime.h"
#include "lb.h"
#include "common.h"

__constant__ int e_d[Q][Dim] = 
{
	{0, 0, 0}, //0

	{1,  0, 0}, //1
	{-1, 0, 0}, //2
	{0,  1, 0}, //3
	{0, -1, 0}, //4
	{0,  0, 1}, //5
	{0,  0,-1}, //6

	{0,  1,  1},//7
	{0, -1, -1},//8
	{0, -1,  1},//9
	{0,  1, -1},//10
	{-1, 0, -1},//11
	{ 1, 0,  1},//12
	{-1, 0,  1},//13
	{ 1, 0, -1},//14
	{-1, 1,  0},//15
	{ 1,-1,  0},//16
	{-1, -1, 0},//17
	{ 1,  1, 0}//18
};

__constant__ int re_d[Q] = 
{
	0,
	2,
	1,
	4,
	3,
	6,
	5,


	8,
	7,
	10,
	9,
	12,
	11,
	14,
	13,
	16,
	15,
	18,
	17
};

__constant__ double w_d[Q] = 
{
	1.0/3,

	1.0/18,
	1.0/18,
	1.0/18,
	1.0/18,
	1.0/18,
	1.0/18,

	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36,
	1.0/36
};

__constant__ double r0 = 1.0/19;
__constant__ double r1 = 5.0/399;
__constant__ double r2 = 1.0/21;
__constant__ double r3 = 11.0/2394;
__constant__ double r4 = 1.0/63;
__constant__ double r5 = 1.0/10;
__constant__ double r6 = 1.0/18;
__constant__ double r7 = 1.0/36;
__constant__ double r8 = 4.0/1197;
__constant__ double r9 = 1.0/252;
__constant__ double r10 = 1.0/40;
__constant__ double r11 = 1.0/12;
__constant__ double r12 = 1.0/4;
__constant__ double r13 = 1.0/72;
__constant__ double r14 = 1.0/24;
__constant__ double r15 = 1.0/8;
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void Evol_flow(double rgama, double *sf_d, double dt, double Fx, double Fy, double Fz, double *f_d, double *F_d)
{
	double P, U, V, W, UVW;
	int tx;
	int bx, by, bz;
	int z, y, x, k;	
	
	double mf[Q];
	double f0[BX];
	double f1[BX], f2[BX], f3[BX], f4[BX], f5[BX], f6[BX];
	double f7[BX], f8[BX], f9[BX], f10[BX], f11[BX], f12[BX], f13[BX], f14[BX], f15[BX], f16[BX], f17[BX], f18[BX];
	__shared__ double F0[BX];
	__shared__ double F1[BX], F2[BX], F3[BX], F4[BX], F5[BX], F6[BX];
	__shared__ double F7[BX], F8[BX], F9[BX], F10[BX], F11[BX], F12[BX], F13[BX], F14[BX], F15[BX], F16[BX], F17[BX], F18[BX];

	tx = threadIdx.x;
    bx = blockIdx.x;
	by = blockIdx.y;
	bz = blockIdx.z;	
	x = N16+bx*BX+tx;
	y = 1+by;
    z = 1+bz;	
	k = GID(z,y,x);


	if(x <= N16+NX1)
	{

		f0[tx]  = f_d[k+0*size]; 
		f1[tx]  = f_d[k+1*size]; 
		f2[tx]  = f_d[k+2*size]; 
		f3[tx]  = f_d[k+3*size]; 
		f4[tx]  = f_d[k+4*size]; 
		f5[tx]  = f_d[k+5*size]; 
		f6[tx]  = f_d[k+6*size]; 
		f7[tx]  = f_d[k+7*size]; 
		f8[tx]  = f_d[k+8*size]; 
		f9[tx]  = f_d[k+9*size]; 
		f10[tx] = f_d[k+10*size]; 
		f11[tx] = f_d[k+11*size]; 
		f12[tx] = f_d[k+12*size]; 
		f13[tx] = f_d[k+13*size]; 
		f14[tx] = f_d[k+14*size]; 
		f15[tx] = f_d[k+15*size]; 
		f16[tx] = f_d[k+16*size]; 
		f17[tx] = f_d[k+17*size]; 
		f18[tx] = f_d[k+18*size]; 
		
		//f-mf///////////////////////////
		mf[0]  = f0[tx] + f1[tx] + f2[tx] + f3[tx] + f4[tx] + f5[tx] + f6[tx] + f7[tx] + f8[tx] + f9[tx] + f10[tx] + f11[tx] + f12[tx] + f13[tx] + f14[tx] + f15[tx] + f16[tx] + f17[tx] + f18[tx]; 
 		mf[1]  = -30.0*f0[tx] - 11.0*f1[tx] - 11.0*f2[tx] - 11.0*f3[tx] - 11.0*f4[tx] - 11.0*f5[tx]
		       	-11.0*f6[tx] + 8.0*f7[tx] + 8.0*f8[tx] + 8.0*f9[tx] + 8.0*f10[tx] + 8.0*f11[tx] + 8.0*f12[tx] + 8.0*f13[tx] + 8.0*f14[tx] + 8.0*f15[tx] + 8.0*f16[tx] + 8.0*f17[tx] + 8.0*f18[tx];
                mf[2]  = 12.0*f0[tx] - 4.0*f1[tx] - 4.0*f2[tx] - 4.0*f3[tx] - 4.0*f4[tx] - 4.0*f5[tx] - 4.0*f6[tx] + f7[tx] + f8[tx] + f9[tx] 
			+ f10[tx] + f11[tx] + f12[tx] + f13[tx] + f14[tx] + f15[tx] + f16[tx] + f17[tx] + f18[tx];
              	mf[3]  = f1[tx] - f2[tx] - f11[tx] + f12[tx] - f13[tx] + f14[tx] - f15[tx] + f16[tx] - f17[tx] + f18[tx];
                mf[4]  = -4.0*f1[tx] + 4.0*f2[tx]  - f11[tx] + f12[tx] - f13[tx] + f14[tx] - f15[tx] + f16[tx] - f17[tx] + f18[tx];
                mf[5]  = f3[tx] - f4[tx] + f7[tx] - f8[tx] - f9[tx] + f10[tx] + f15[tx] - f16[tx] - f17[tx] + f18[tx];
                mf[6]  = -4.0*f3[tx] + 4.0*f4[tx] + f7[tx] - f8[tx] - f9[tx] + f10[tx] + f15[tx] - f16[tx] - f17[tx] + f18[tx];
                mf[7]  = f5[tx] - f6[tx] + f7[tx] - f8[tx] + f9[tx] + f12[tx] - f11[tx] - f10[tx] + f13[tx] - f14[tx];
                mf[8]  = f12[tx] - f11[tx] - f10[tx] + f13[tx] - f14[tx] - 4.0*f5[tx] + 4.0*f6[tx] + f7[tx] - f8[tx] + f9[tx];
                mf[9]  = 2.0*f1[tx] - 2.0*f10[tx] + f11[tx] + f12[tx] + f13[tx] + f14[tx] + f15[tx] + f16[tx] + f17[tx] + f18[tx] + 2.0*f2[tx] - f3[tx] - f4[tx] - f5[tx] - f6[tx] - 2.0*f7[tx] - 2.0*f8[tx] - 2.0*f9[tx];
                mf[10] = f11[tx] - 2.0*f10[tx] - 4.0*f1[tx] + f12[tx] + f13[tx] + f14[tx]
		       	+ f15[tx] + f16[tx] + f17[tx] + f18[tx] - 4.0*f2[tx] + 2.0*f3[tx] + 2.0*f4[tx] + 2.0*f5[tx] + 2.0*f6[tx] - 2.0*f7[tx] - 2.0*f8[tx] - 2.0*f9[tx];
                mf[11] = f15[tx] - f12[tx] - f13[tx] - f14[tx] - f11[tx] + f16[tx] + f17[tx] + f18[tx] + f3[tx] + f4[tx] - f5[tx] - f6[tx];
                mf[12] = f15[tx] - f12[tx] - f13[tx] - f14[tx] - f11[tx] + f16[tx] + f17[tx] + f18[tx] - 2.0*f3[tx] - 2.0*f4[tx] + 2.0*f5[tx] + 2.0*f6[tx];
                mf[13] = f17[tx] - f16[tx] - f15[tx] + f18[tx];
                mf[14] = f7[tx] - f10[tx] + f8[tx] - f9[tx];
                mf[15] = f11[tx] + f12[tx] - f13[tx] - f14[tx];
                mf[16] = f11[tx] - f12[tx] + f13[tx] - f14[tx] - f15[tx] + f16[tx] - f17[tx] + f18[tx];
                mf[17] = f10[tx] - f15[tx] + f16[tx] + f17[tx] - f18[tx] + f7[tx] - f8[tx] - f9[tx];
                mf[18] = f10[tx] - f11[tx] + f12[tx] + f13[tx] - f14[tx] - f7[tx] + f8[tx] - f9[tx];

		//macroscopic variables/////////////////////////////////////////////////////////////////////////////
		U   = f1[tx] + f12[tx] + f14[tx] + f16[tx] + f18[tx] - f2[tx] - f11[tx] - f13[tx] - f15[tx] - f17[tx] + 0.5*dt*Fx;
		V   = f3[tx] + f7[tx]  + f10[tx] + f15[tx] + f18[tx] - f4[tx] - f8[tx]  - f9[tx]  - f16[tx] - f17[tx] + 0.5*dt*Fy;
		W   = f5[tx] + f7[tx]  + f9[tx]  + f12[tx] + f13[tx] - f6[tx] - f8[tx]  - f10[tx] - f11[tx] - f14[tx] + 0.5*dt*Fz;
		UVW = U*U + V*V + W*W;
		P = (f0[tx] + f1[tx] + f2[tx] + f3[tx] + f4[tx] + f5[tx] + f6[tx] + f7[tx] + f8[tx] + f9[tx] + f10[tx] + f11[tx] + f12[tx] + f13[tx] 
				+ f14[tx] + f15[tx] + f16[tx] + f17[tx] + f18[tx]);
		
		//collision//-------------------------------------------------------------------------------------------------------------------------------
		mf[0]  = (mf[0] - sf_d[0]*( mf[0] - MEQ_0(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[0])*F_0(U,V,W,Fx,Fy,Fz,rgama);
		mf[1]  = (mf[1] - sf_d[1]*( mf[1] - MEQ_1(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[1])*F_1(U,V,W,Fx,Fy,Fz,rgama);
		mf[2]  = (mf[2] - sf_d[2]*( mf[2] - MEQ_2(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[2])*F_2(U,V,W,Fx,Fy,Fz,rgama);
		mf[3]  = (mf[3] - sf_d[3]*( mf[3] - MEQ_3(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[3])*F_3(U,V,W,Fx,Fy,Fz,rgama);
		mf[4]  = (mf[4] - sf_d[4]*( mf[4] - MEQ_4(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[4])*F_4(U,V,W,Fx,Fy,Fz,rgama);
		mf[5]  = (mf[5] - sf_d[5]*( mf[5] - MEQ_5(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[5])*F_5(U,V,W,Fx,Fy,Fz,rgama);
		mf[6]  = (mf[6] - sf_d[6]*( mf[6] - MEQ_6(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[6])*F_6(U,V,W,Fx,Fy,Fz,rgama);
		mf[7]  = (mf[7] - sf_d[7]*( mf[7] - MEQ_7(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[7])*F_7(U,V,W,Fx,Fy,Fz,rgama);
		mf[8]  = (mf[8] - sf_d[8]*( mf[8] - MEQ_8(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[8])*F_8(U,V,W,Fx,Fy,Fz,rgama);
		mf[9]  = (mf[9] - sf_d[9]*( mf[9] - MEQ_9(P, U, V, W, rgama)))+dt*(1.0 - 0.5*sf_d[9])*F_9(U,V,W,Fx,Fy,Fz,rgama);
		mf[10] = (mf[10]-sf_d[10]*(mf[10] -MEQ_10(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[10])*F_10(U,V,W,Fx,Fy,Fz,rgama);
		mf[11] = (mf[11]-sf_d[11]*(mf[11] -MEQ_11(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[11])*F_11(U,V,W,Fx,Fy,Fz,rgama);
		mf[12] = (mf[12]-sf_d[12]*(mf[12] -MEQ_12(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[12])*F_12(U,V,W,Fx,Fy,Fz,rgama);
		mf[13] = (mf[13]-sf_d[13]*(mf[13] -MEQ_13(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[13])*F_13(U,V,W,Fx,Fy,Fz,rgama);
		mf[14] = (mf[14]-sf_d[14]*(mf[14] -MEQ_14(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[14])*F_14(U,V,W,Fx,Fy,Fz,rgama);
		mf[15] = (mf[15]-sf_d[15]*(mf[15] -MEQ_15(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[15])*F_15(U,V,W,Fx,Fy,Fz,rgama);
		mf[16] = (mf[16]-sf_d[16]*(mf[16] -MEQ_16(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[16])*F_16(U,V,W,Fx,Fy,Fz,rgama);
		mf[17] = (mf[17]-sf_d[17]*(mf[17] -MEQ_17(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[17])*F_17(U,V,W,Fx,Fy,Fz,rgama);
		mf[18] = (mf[18]-sf_d[18]*(mf[18] -MEQ_18(P, U, V, W, rgama)))+dt*(1.0 -0.5*sf_d[18])*F_18(U,V,W,Fx,Fy,Fz,rgama);

		//--mf - f --//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		F0[tx]  = r0*mf[0] - r1*mf[1] + r2*mf[2];
                F1[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[3] - r5*mf[4] + r6*mf[9] - r6*mf[10];
                F2[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[3] + r5*mf[4] + r6*mf[9] - r6*mf[10]; 
                F3[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[5] - r5*mf[6] - r7*mf[9] + r7*mf[10] + r11*mf[11] - r11*mf[12]; 
                F4[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[5] + r5*mf[6] - r7*mf[9] + r7*mf[10] + r11*mf[11] - r11*mf[12];
                F5[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] + r5*mf[7] - r5*mf[8] - r7*mf[9] + r7*mf[10] - r11*mf[11] + r11*mf[12];
                F6[tx]  = r0*mf[0] - r3*mf[1] - r4*mf[2] - r5*mf[7] + r5*mf[8] - r7*mf[9] + r7*mf[10] - r11*mf[11] + r11*mf[12];
                F7[tx]  = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[5] + r10*mf[6] + r5*mf[7] + r10*mf[8] - r6*mf[9]  - r7*mf[10] + r12*mf[14] + r15*mf[17] - r15*mf[18];
                F8[tx]  = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[5] - r10*mf[6] - r5*mf[7] - r10*mf[8] - r6*mf[9]  - r7*mf[10] + r12*mf[14] - r15*mf[17] + r15*mf[18];
                F9[tx]  = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[5] - r10*mf[6] + r5*mf[7] + r10*mf[8] - r6*mf[9]  - r7*mf[10] - r12*mf[14] - r15*mf[17] - r15*mf[18];
                F10[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[5] + r10*mf[6] - r5*mf[7] - r10*mf[8] - r6*mf[9]  - r7*mf[10] - r12*mf[14] + r15*mf[17] + r15*mf[18];
                F11[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] - r5*mf[7] - r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] + r12*mf[15] + r15*mf[16] - r15*mf[18];
                F12[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] + r5*mf[7] + r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] + r12*mf[15] - r15*mf[16] + r15*mf[18];
                F13[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] + r5*mf[7] + r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] - r12*mf[15] + r15*mf[16] + r15*mf[18]; 
                F14[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] - r5*mf[7] - r10*mf[8] + r7*mf[9] + r13*mf[10] - r11*mf[11] - r14*mf[12] - r12*mf[15] - r15*mf[16] - r15*mf[18]; 
                F15[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] + r5*mf[5] + r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] - r12*mf[13] - r15*mf[16] - r15*mf[17]; 
                F16[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] - r5*mf[5] - r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] - r12*mf[13] + r15*mf[16] + r15*mf[17]; 
                F17[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] - r5*mf[3] - r10*mf[4] - r5*mf[5] - r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] + r12*mf[13] - r15*mf[16] + r15*mf[17];
                F18[tx] = r0*mf[0] + r8*mf[1] + r9*mf[2] + r5*mf[3] + r10*mf[4] + r5*mf[5] + r10*mf[6] + r7*mf[9] + r13*mf[10] + r11*mf[11] + r14*mf[12] + r12*mf[13] + r15*mf[16] - r15*mf[17];
		
		__syncthreads();

		//  streaming  ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		F_d[k           + 0*size] = F0[tx];


		F_d[k + NX2     + 3*size] = F3[tx];
		F_d[k - NX2     + 4*size] = F4[tx];
		F_d[k + NX2*NY2 + 5*size] = F5[tx];
		F_d[k - NX2*NY2 + 6*size] = F6[tx];


		F_d[k + NX2 + NX2*NY2 + 7*size] = F7[tx];
		F_d[k - NX2 - NX2*NY2 + 8*size] = F8[tx];
		F_d[k - NX2 + NX2*NY2 + 9*size] = F9[tx];
		F_d[k + NX2 - NX2*NY2 +10*size] = F10[tx];

		if(tx != 0)
		{
			F_d[k           +  1*size] =  F1[tx-1];
			F_d[k + NX2*NY2 + 12*size] = F12[tx-1];
			F_d[k - NX2*NY2 + 14*size] = F14[tx-1];
			F_d[k - NX2     + 16*size] = F16[tx-1];
			F_d[k + NX2     + 18*size] = F18[tx-1];
		}

		if(tx == BX-1)
		{
			F_d[k + 1           +  1*size] = F1[tx];
			F_d[k + 1 + NX2*NY2 + 12*size] = F12[tx];
			F_d[k + 1 - NX2*NY2 + 14*size] = F14[tx];
			F_d[k + 1 - NX2     + 16*size] = F16[tx];
			F_d[k + 1 + NX2     + 18*size] = F18[tx];
		}

		if(tx != BX-1)
		{	
			F_d[k           +  2*size] = F2[tx+1];
			F_d[k - NX2*NY2 + 11*size] = F11[tx+1];
			F_d[k + NX2*NY2 + 13*size] = F13[tx+1];
			F_d[k + NX2     + 15*size] = F15[tx+1];
			F_d[k - NX2     + 17*size] = F17[tx+1];
		}

		if(tx == 0)
		{
			F_d[k - 1           +  2*size] = F2[tx];
			F_d[k - 1 - NX2*NY2 + 11*size] = F11[tx];
			F_d[k - 1 + NX2*NY2 + 13*size] = F13[tx];
			F_d[k - 1 + NX2     + 15*size] = F15[tx];
			F_d[k - 1 - NX2     + 17*size] = F17[tx];        
		}
	}
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Bc_flow_BB(int *flag_d, double *F_d)
{
	int tx, bx, by, bz;
	int z, y, x, k;
	int zp, yp, xp, kp;
	int q;


	tx = threadIdx.x;
     	bx = blockIdx.x;
	by = blockIdx.y;
	bz = blockIdx.z;	
	x = N16+bx*BX+tx;
	y = 1+by;
       	z = 1+bz;	
	k = GID(z,y,x);

	if(x < N16+NX1)
	{
		if(flag_d[k] == 0)
		{
			for(q = 1; q < Q; q++)
			{
				xp = x - e_d[q][0]; yp = y - e_d[q][1]; zp = z - e_d[q][2];
				kp = GID(zp,yp,xp);
				if(flag_d[kp] == 1)
				{
					F_d[k + q*size] = F_d[kp + re_d[q]*size];
				}

			}
		}
	}

}

//-----------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Bc_flow_io(double *F_d)       
{
	int ty, bx, bz;
	int z, y, x, k;
	int zp, yp, xp, kp;

	ty = threadIdx.x; 
	bx = blockIdx.x;
	bz = blockIdx.y;
	y = 1 + bx*BCX + ty;
	z = 1 + bz;

	if(y <= NY1)
	{
		//inlet
		x = N16;
		k = GID(z, y, x);
		// c_1
		xp = N16+NX1;
		yp = y; zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 1*size] = F_d[kp + 1*size];

		// c_12
		yp = y;
       		if(z > 1) zp = z;
		else zp = NZ1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 12*size] = F_d[kp + 12*size];

		// c_14
		yp = y;
		if(z < NZ1) zp = z;
		else zp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 14*size] = F_d[kp + 14*size];

		// c_16
		if(y < NY1) yp = y;
		else yp = 0; 
		zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 16*size] = F_d[kp + 16*size];

		// c_18
		if(y > 1) yp = y;
		else yp = NY1+1; 
		zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 18*size] = F_d[kp + 18*size];

		//outlet
		x = N16+NX;
		k = GID(z, y, x);
		// c_2
		xp = N16-1;
		yp = y; zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 2*size] = F_d[kp + 2*size];

		// c_11
		yp = y;
       		if(z < NZ1) zp = z;
		else zp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 11*size] = F_d[kp + 11*size];

		// c_13
		yp = y;
		if(z > 1) zp = z;
		else zp = NZ1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 13*size] = F_d[kp + 13*size];

		// c_15
		if(y > 1) yp = y;
		else yp = NY1+1; 
		zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 15*size] = F_d[kp + 15*size];

		// c_17
		if(y < NY1) yp = y;
		else yp = 0; 
		zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 17*size] = F_d[kp + 17*size];	
	}
}
//------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Bc_flow_fb(double *F_d)       
{
	int tx, bx, bz;
	int z, y, x, k;
	int zp, yp, xp, kp;

	tx = threadIdx.x; 
	bx = blockIdx.x;
	bz = blockIdx.y;
	x = N16 + bx*BCX + tx;
	z = 1 + bz;

	if(x < N16+NX1)
	{
		//front
		y = 1;
		k = GID(z, y, x);
		// c_3
		yp = NY1+1;
		xp = x; zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 3*size] = F_d[kp + 3*size];

		// c_7
		xp = x;
      		if(z > 1) zp = z;
		else zp = NZ1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 7*size] = F_d[kp + 7*size];

		// c_10
		xp = x;
		if(z < NZ1) zp = z;
		else zp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 10*size] = F_d[kp + 10*size];

		// c_15
		zp = z;
		if(x < N16+NX) xp = x;
		else xp = N16-1;
		kp = GID(zp, yp, xp);
		F_d[k + 15*size] = F_d[kp + 15*size];

		// c_18
		zp = z;
		if(x > N16) xp = x;
		else xp = N16+NX1;
		kp = GID(zp, yp, xp);
		F_d[k + 18*size] = F_d[kp + 18*size];

		//back
		y = NY1;
		k = GID(z, y, x);
		// c_4
		yp = 0;
		xp = x; zp = z;
		kp = GID(zp, yp, xp);
		F_d[k + 4*size] = F_d[kp + 4*size];

		// c_8
		xp = x;
    		if(z < NZ1) zp = z;
		else zp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 8*size] = F_d[kp + 8*size];

		// c_9
		xp = x;
		if(z > 1) zp = z;
		else zp = NZ1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 9*size] = F_d[kp + 9*size];

		// c_16
		zp = z;
		if(x > N16) xp = x;
		else xp = N16+NX1;
		kp = GID(zp, yp, xp);
		F_d[k + 16*size] = F_d[kp + 16*size];

		// c_17
		zp = z;
		if(x < N16+NX) xp = x;
		else xp = N16-1;
		kp = GID(zp, yp, xp);
		F_d[k + 17*size] = F_d[kp + 17*size];
	}
	

}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
__global__ void Bc_flow_ub(double *F_d)       
{
	int tx, bx, by;
	int z, y, x, k;
	int zp, yp, xp, kp;

	tx = threadIdx.x; 
	bx = blockIdx.x;
	by = blockIdx.y;
	x = N16 + bx*BCX + tx;
	y = 1 + by;

	if(x < N16+NX1)
	{
		//bottom
		z = 1;
		k = GID(z, y, x);
		// c_5
		zp = NZ1+1;
		xp = x; yp = y;
		kp = GID(zp, yp, xp);
		F_d[k + 5*size] = F_d[kp + 5*size];

		// c_7
		xp = x; 
		if(y > 1) yp = y;
		else yp = NY1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 7*size] = F_d[kp + 7*size];

		// c_9
		xp = x; 
		if(y < NY1) yp = y;
		else yp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 9*size] = F_d[kp + 9*size];

		// c_12
		yp = y; 
		if(x > N16) xp = x;
		else xp = N16+NX1;
		kp = GID(zp, yp, xp);
		F_d[k + 12*size] = F_d[kp + 12*size];

		// c_13
		yp = y; 
		if(x < N16+NX) xp = x;
		else xp = N16-1;
		kp = GID(zp, yp, xp);
		F_d[k + 13*size] = F_d[kp + 13*size];


		//upper
		z = NZ1;
		k = GID(z, y, x);
		// c_6
		zp = 0;
		yp = y; xp = x;
		kp = GID(zp, yp, xp);
		F_d[k + 6*size] = F_d[kp + 6*size];

		// c_8
		xp = x; 
		if(y < NY1) yp = y;
		else yp = 0;
		kp = GID(zp, yp, xp);
		F_d[k + 8*size] = F_d[kp + 8*size];

		// c_10
		xp = x; 
		if(y > 1) yp = y;
		else yp = NY1+1;
		kp = GID(zp, yp, xp);
		F_d[k + 10*size] = F_d[kp + 10*size];

		// c_11
		yp = y; 
		if(x < N16+NX) xp = x;
		else xp = N16-1;
		kp = GID(zp, yp, xp);
		F_d[k + 11*size] = F_d[kp + 11*size];

		// c_14
		yp = y; 
		if(x > N16) xp = x;
		else xp = N16+NX1;
		kp = GID(zp, yp, xp);
		F_d[k + 14*size] = F_d[kp + 14*size];
	}

}
