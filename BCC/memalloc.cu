#include <stdlib.h>
#include <malloc.h>
#include "common.h"
#include "lb.h"

void memalloc()
{
    //allocate memeory at host
    f    = (double*)calloc(size*Q, sizeof(double));
    nc   = (double*)calloc(NR*Dim, sizeof(double));
    sf   = (double*)calloc(Q, sizeof(double));
    flag = (int*)calloc(size, sizeof(int));

    //allocate memeory at device
    hipMalloc((void **)&f_dev, size*Q*sizeof(double));
    hipMalloc((void **)&F_dev, size*Q*sizeof(double));
    hipMalloc((void **)&sf_dev, Q*sizeof(double));
    hipMalloc((void **)&flag_dev, size*sizeof(int));
}

