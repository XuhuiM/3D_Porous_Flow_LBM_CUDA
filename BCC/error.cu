#include "hip/hip_runtime.h"
#include "lb.h"
#include "common.h"

double error()
{
	int z, y, x, k;
	int q;
	double ut;
	double sum_u_c = 0.0;
	double Err;
	FILE *fp;

	for(z = 1; z <= NZ1; z++)
		for(y = 1; y <= NY1; y++)
			for(x = N16; x < N16+NX1; x++)
			{
				k = GID(z,y,x);
				if(flag[k] == 0)
				{
					ut = 0.0;
					for(q = 1; q < Q; q++)
					{
						ut += e[q][0]*f[k+q*size];
					}	
					ut += 0.5*dt*Fx;
				}
				sum_u_c += ut;
			}
	

	Err = fabs(sum_u_c - sum_u_o)/fabs(sum_u_c);
	sum_u_o = sum_u_c;

	if((fp=fopen("err.dat", "a"))==NULL) printf("FILE OPEN ERROR!\n");
	fprintf(fp, "%d %e\n", n, Err);
	fclose(fp);

	return Err;

}
