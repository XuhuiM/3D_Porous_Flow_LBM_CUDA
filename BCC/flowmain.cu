#include "hip/hip_runtime.h"
//---LBE for 3D porous flow----------------------------------------------------------
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include <time.h>
#include "lb.h"
#include "common.h"
#include "memalloc.cu"
#include "geo.cu"
#include "init.cu"
#include "flow.cu"
#include "error.cu"
#include "datasave.cu"
//----------------------------------------------------------------------------------------------------------------------------
int main()
{
	int new_step, goon;
	double err = 1.0;
	clock_t time_begin, time_end;

	int device = 0;
    	hipSetDevice(device);
    	hipDeviceProp_t properties;
    	hipGetDeviceProperties(&properties, device);
    	printf("Lattice Boltzmann Simulation running on: %s\n", properties.name);

	dim3 threads(BX, 1, 1);
  	dim3 threads_BC(BCX, 1, 1);	
	dim3 grid((NX1+BX-1)/BX, NY1, NZ1);
   	dim3 grid_BC_io((NY1+BCX-1)/BCX, NZ1);
	dim3 grid_BC_fb((NX1+BCX-1)/BCX, NZ1);
	dim3 grid_BC_ub((NX1+BCX-1)/BCX, NY1);
	
	srand((unsigned) time(NULL));

	memalloc();
	geo();
	LB_init();
	datasave();
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipMemcpy(f_dev,  f, sizeof(double)*Q*size, hipMemcpyHostToDevice);
	hipMemcpy(flag_dev,  flag, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(sf_dev, sf, sizeof(double)*Q, hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////o///////////////////////////////////////////////////////

	time_begin = clock();
loop:
	printf("Enter the num of steps:");
    	scanf("%d", &new_step);
	nmax += new_step;
	printf("nmax = %d\n", nmax);

	while((n < nmax)&&(err > 1.0e-6))
	{
	
		n++;
		if(n%2 == 0)
		{
			fold = F_dev;
			fnew = f_dev;
		}
		else
		{
			fold = f_dev;
			fnew = F_dev;
		}

		Evol_flow<<< grid, threads >>>(rgama, sf_dev, dt, Fx, Fy, Fz, fold, fnew);
		Bc_flow_io<<< grid_BC_io, threads_BC >>>(fnew);
		Bc_flow_fb<<< grid_BC_fb, threads_BC >>>(fnew);
		Bc_flow_ub<<< grid_BC_ub, threads_BC >>>(fnew);
		Bc_flow_BB<<< grid, threads >>>(flag_dev, fnew);
	
		if(n%TP == 0)
		{
		       hipMemcpy(f, f_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost);	
		       err = error();		
		       printf( "n=%d: err = %e\n", n, err);
		       if(n%10000 == 0) datasave();
		}

	}

	time_end = clock();
	printf( "The computing time is: %f mins \n", (double)( time_end - time_begin ) / CLOCKS_PER_SEC / 60.f );
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	if(n % 2 == 0)
	{
		hipMemcpy( f, f_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost );
		printf("this is from f\n");
	}
	else
	{
	    hipMemcpy( f, F_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost );
	    printf("this is from F\n");
	}

	datasave();

	printf("goon? yes(1) no(0):");
	scanf("%d", &goon);
	if(goon) goto loop;

	
	hipFree(f_dev);
	hipFree(F_dev);
	hipFree(flag_dev);
	hipFree(sf_dev);
	////////////////////////////////////////////////////////////////////////////////////////////////_GPU
	return 0;

}
