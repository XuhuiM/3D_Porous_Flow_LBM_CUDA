#include "hip/hip_runtime.h"
#include "lb.h"
#include "common.h"

void LB_init()
{
	int k;
	double NX;
	double ut, vt, wt, pt;
	double G;

	NX = 90;
	Lx = 1.0;
	Ly = Lx;
	Lz = Lx;
	gama = 1.0;
	rgama = 1.0/gama;
	dx = Lx/NX;
	dt = dx;
	ci = dx/dt;
	rcc = 3.0/ci/ci;
	tau_f = 1.0;
	wf = 1.0/tau_f;
	nu = (tau_f - 0.5)*dt/3;
	U0 = Re*nu/Ly;
//	G = U0*nu*PI*PI*PI/4;
	G = 1.0e-5;
	Fx = G*rgama;
	Fy = 0.0;
	Fz = 0.0;


	sf[0] = 0.0;
	sf[1] = wf; sf[2] = wf;
	sf[3] = sf[5] = sf[7] = wf;
	sf[9] = sf[11] = sf[13] = sf[14] = sf[15] = wf;
	sf[4] = sf[6] = sf[8] = (16.0*tau_f - 8.0)/(8.0*tau_f - 1.0);
	sf[10] = sf[12] = wf;
	sf[16] = sf[17] = sf[18] = sf[4];
	 
	
	printf("tau_f = %f, Ma = %f, G = %e\n", tau_f, sqrt(3.0*rgama)*U0/ci, G);

	ut = 0.0; 
	vt = wt = 0.0;
	pt = 1.0;
	for(k = 0; k < N; k++)
	{
		f[k+0*size]  = FEQ_0(pt,ut,vt,wt,rgama);
		f[k+1*size]  = FEQ_1(pt,ut,vt,wt,rgama);
		f[k+2*size]  = FEQ_2(pt,ut,vt,wt,rgama);
		f[k+3*size]  = FEQ_3(pt,ut,vt,wt,rgama);
		f[k+4*size]  = FEQ_4(pt,ut,vt,wt,rgama);
		f[k+5*size]  = FEQ_5(pt,ut,vt,wt,rgama);
		f[k+6*size]  = FEQ_6(pt,ut,vt,wt,rgama);
		f[k+7*size]  = FEQ_7(pt,ut,vt,wt,rgama);
		f[k+8*size]  = FEQ_8(pt,ut,vt,wt,rgama);
		f[k+9*size]  = FEQ_9(pt,ut,vt,wt,rgama);

		f[k+10*size] = FEQ_10(pt,ut,vt,wt,rgama);
		f[k+11*size] = FEQ_11(pt,ut,vt,wt,rgama);
		f[k+12*size] = FEQ_12(pt,ut,vt,wt,rgama);
		f[k+13*size] = FEQ_13(pt,ut,vt,wt,rgama);
		f[k+14*size] = FEQ_14(pt,ut,vt,wt,rgama);
		f[k+15*size] = FEQ_15(pt,ut,vt,wt,rgama);
		f[k+16*size] = FEQ_16(pt,ut,vt,wt,rgama);
		f[k+17*size] = FEQ_17(pt,ut,vt,wt,rgama);
		f[k+18*size] = FEQ_18(pt,ut,vt,wt,rgama);

	}	
}
