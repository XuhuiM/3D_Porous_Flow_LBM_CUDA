#include "hip/hip_runtime.h"
//---LBE for 3D porous flow----------------------------------------------------------
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include <time.h>
#include "lb.h"
#include "common.h"
#include "memalloc.cu"
#include "geo.cu"
#include "init.cu"
#include "flow.cu"
#include "error.cu"
#include "datasave.cu"
//----------------------------------------------------------------------------------------------------------------------------
int main()
{
	unsigned int new_step, goon;
	hipEvent_t start, stop;
	float GPU_Time;	
	double err = 1.0;
	clock_t time_begin, time_end;

	int device = 1;
    hipSetDevice(device);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);
    printf("Lattice Boltzmann Simulation running on: %s\n", properties.name);

	memalloc();

	dim3 threads(BX, 1, 1);
	dim3 grid((N+BX-1)/BX, 1, 1);
	
	geo();
	LB_init();
	datasave();	
	///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	hipMemcpy(f_dev,  f, sizeof(double)*Q*size, hipMemcpyHostToDevice);
	hipMemcpy(sf_dev, sf, sizeof(double)*Q, hipMemcpyHostToDevice);
	hipMemcpy(node_dev,  node_index, sizeof(int)*Q*size, hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////o///////////////////////////////////////////////////////

loop:
	printf("Enter the num of steps:");
    scanf("%u", &new_step);
	nmax += new_step;
	printf("nmax = %u\n", nmax);

	time_begin = clock();
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);	
	while((n < nmax)&&(err > 1.0e-6))
	{
	
		n++;
		if(n%2 == 0)
		{
			fold = F_dev;
			fnew = f_dev;
		}
		else
		{
			fold = f_dev;
			fnew = F_dev;
		}

		Evol_flow<<< grid, threads >>>(rgama, sf_dev, dt, Fx, Fy, Fz, N, size, node_dev, fold, fnew);
	
		if(n%TP == 0)
		{
//		       hipMemcpy(f, f_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost);	
//		       err = error();		
		       printf( "n=%u: err = %e\n", n, err);
//		       datasave();
		}

	}

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&GPU_Time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	time_end = clock();
	printf("The computing time is: %f s, %f mins\n", (double)( time_end - time_begin ) / CLOCKS_PER_SEC, (double)( time_end - time_begin ) / CLOCKS_PER_SEC / 60.f);
	printf("MUPLS (s) for GPU is %f\n", 1.0*size*n/1000000.0/GPU_Time*1000.0);
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	if(n % 2 == 0)
	{
		hipMemcpy(f, f_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost);
		printf("this is from f\n");
	}
	else
	{
	    hipMemcpy(f, F_dev, Q*size*sizeof(double), hipMemcpyDeviceToHost);
	    printf("this is from F\n");
	}

	datasave();

	printf("goon? yes(1) no(0):");
	scanf("%u", &goon);
	if(goon) goto loop;
	
	free(f);
	free(node_index);
	free(sf);
	hipFree(f_dev);
	hipFree(F_dev);
	hipFree(node_dev);
	hipFree(sf_dev);
	////////////////////////////////////////////////////////////////////////////////////////////////_GPU
	return 0;

}
